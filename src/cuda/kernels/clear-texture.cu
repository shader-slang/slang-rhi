// This file implements CUDA kernels for clearing textures.
// The kernels are named clearTexture{dim}_{bits}_{layered}, where:
// - {dim} is the dimension of the texture (1D, 2D, 3D, Cube)
// - {bits} is the number of bits per element (8, 16, 32, 64, 128)
// - {layered} is 0 if the texture is not layered, 1 if it is layered
// The kernels are called with the following arguments:
// - surface: the surface object of the texture to clear
// - sizeAndLayer: a uint4 containing the size of the texture (width, height, depth) and the layer index
// - value: a uint4 containing the raw value to write to the texture
//
// To recompile this file into the corresponding PTX code, run:
// nvcc -ptx -o clear-texture.ptx clear-texture.cu

#include <hip/hip_runtime.h>

template<typename T, bool Layered>
__device__ void clearTexture1D(hipSurfaceObject_t surface, uint4 sizeAndLayer, uint4 value)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < sizeAndLayer.x)
    {
        T tmp = *((T*)&value);
        if (Layered)
            surf1DLayeredwrite(tmp, surface, x * sizeof(T), sizeAndLayer.w);
        else
            surf1Dwrite(tmp, surface, x * sizeof(T));
    }
}

template<typename T, bool Layered>
__device__ void clearTexture2D(hipSurfaceObject_t surface, uint4 sizeAndLayer, uint4 value)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < sizeAndLayer.x && y < sizeAndLayer.y)
    {
        T tmp = *((T*)&value);
        if (Layered)
            surf2DLayeredwrite(tmp, surface, x * sizeof(T), y, sizeAndLayer.w);
        else
            surf2Dwrite(tmp, surface, x * sizeof(T), y);
    }
}

template<typename T, bool Layered>
__device__ void clearTexture3D(hipSurfaceObject_t surface, uint4 sizeAndLayer, uint4 value)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < sizeAndLayer.x && y < sizeAndLayer.y && z < sizeAndLayer.z)
    {
        T tmp = *((T*)&value);
        surf3Dwrite(tmp, surface, x * sizeof(T), y, z);
    }
}

template<typename T, bool Layered>
__device__ void clearTextureCube(hipSurfaceObject_t surface, uint4 sizeAndLayer, uint4 value)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < sizeAndLayer.x && y < sizeAndLayer.y)
    {
        T tmp = *((T*)&value);
        if (Layered)
        {
            surfCubemapLayeredwrite(tmp, surface, x * sizeof(T), y, sizeAndLayer.w);
        }
        else
        {
            surfCubemapwrite(tmp, surface, x * sizeof(T), y, sizeAndLayer.w);
        }
    }
}

#define DEFINE_SINGLE(dim, type, bits, layered)                                                                        \
    extern "C" __global__ void clearTexture##dim##_##bits##_##layered(                                                 \
        hipSurfaceObject_t surface,                                                                                   \
        uint4 sizeAndLayer,                                                                                            \
        uint4 value                                                                                                    \
    )                                                                                                                  \
    {                                                                                                                  \
        clearTexture##dim<type, layered>(surface, sizeAndLayer, value);                                                \
    }

#define DEFINE_ALL(type, bits)                                                                                         \
    DEFINE_SINGLE(1D, type, bits, 0)                                                                                   \
    DEFINE_SINGLE(1D, type, bits, 1)                                                                                   \
    DEFINE_SINGLE(2D, type, bits, 0)                                                                                   \
    DEFINE_SINGLE(2D, type, bits, 1)                                                                                   \
    DEFINE_SINGLE(3D, type, bits, 0)                                                                                   \
    DEFINE_SINGLE(Cube, type, bits, 0)                                                                                 \
    DEFINE_SINGLE(Cube, type, bits, 1)

DEFINE_ALL(char1, 8)
DEFINE_ALL(short1, 16)
DEFINE_ALL(int1, 32)
DEFINE_ALL(int2, 64)
DEFINE_ALL(int4, 128)
